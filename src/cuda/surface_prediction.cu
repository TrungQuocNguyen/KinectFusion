#include "hip/hip_runtime.h"
#include <cuda/kernel_common.cuh>
#include <datatypes.hpp>


__device__ __forceinline__
float interpolate_trilinearly(
    const Vector3f_da& point, const PtrStep<short2>& volume,
    const int3& volume_size, const float& voxel_scale
) 
{
    Vector3i_da point_in_grid = point.cast<int>();

    const float vx = (static_cast<float>(point_in_grid[0]) + 0.5f);
    const float vy = (static_cast<float>(point_in_grid[1]) + 0.5f);
    const float vz = (static_cast<float>(point_in_grid[2]) + 0.5f);

    point_in_grid.x() = (point[0] < vx) ? (point_in_grid[0] - 1) : point_in_grid[0];
    point_in_grid.y() = (point[1] < vy) ? (point_in_grid[1] - 1) : point_in_grid[1];
    point_in_grid.z() = (point[2] < vz) ? (point_in_grid[2] - 1) : point_in_grid[2];

    const float a = point.x() - (static_cast<float>(point_in_grid.x()) + 0.5f);
    const float b = point.y() - (static_cast<float>(point_in_grid.y()) + 0.5f);
    const float c = point.z() - (static_cast<float>(point_in_grid.z()) + 0.5f);

    const short2* v0 = volume.ptr((point_in_grid[2]) * volume_size.y + point_in_grid[1]);
    const short2* v1 = volume.ptr((point_in_grid[2] + 1) * volume_size.y + point_in_grid[1]);
    const short2* v2 = volume.ptr((point_in_grid[2]) * volume_size.y + point_in_grid[1] + 1);
    const short2* v3 = volume.ptr((point_in_grid[2] + 1) * volume_size.y + point_in_grid[1] + 1);

    return 
        static_cast<float>(v0[point_in_grid[0]].x) * INV_SHORT_MAX * (1 - a) * (1 - b) * (1 - c) +
        static_cast<float>(v1[point_in_grid[0]].x) * INV_SHORT_MAX * (1 - a) * (1 - b) * c +
        static_cast<float>(v2[point_in_grid[0]].x) * INV_SHORT_MAX * (1 - a) * b * (1 - c) +
        static_cast<float>(v3[point_in_grid[0]].x) * INV_SHORT_MAX * (1 - a) * b * c +
        static_cast<float>(v0[point_in_grid[0] + 1].x) * INV_SHORT_MAX * a * (1 - b) * (1 - c) +
        static_cast<float>(v1[point_in_grid[0] + 1].x) * INV_SHORT_MAX * a * (1 - b) * c +
        static_cast<float>(v2[point_in_grid[0] + 1].x) * INV_SHORT_MAX * a * b * (1 - c) +
        static_cast<float>(v3[point_in_grid[0] + 1].x) * INV_SHORT_MAX * a * b * c;
}


__device__ __forceinline__ float get_min_time(const float3& volume_max, const Vector3f_da& origin, const Vector3f_da& direction)
{
    float txmin = ((direction.x() > 0 ? 0.f : volume_max.x) - origin.x()) / direction.x();
    float tymin = ((direction.y() > 0 ? 0.f : volume_max.y) - origin.y()) / direction.y();
    float tzmin = ((direction.z() > 0 ? 0.f : volume_max.z) - origin.z()) / direction.z();

    return fmax(fmax(txmin, tymin), tzmin);
}


__device__ __forceinline__ float get_max_time(const float3& volume_max, const Vector3f_da& origin, const Vector3f_da& direction)
{
    float txmax = ((direction.x() > 0 ? volume_max.x : 0.f) - origin.x()) / direction.x();
    float tymax = ((direction.y() > 0 ? volume_max.y : 0.f) - origin.y()) / direction.y();
    float tzmax = ((direction.z() > 0 ? volume_max.z : 0.f) - origin.z()) / direction.z();

    return fmin(fmin(txmax, tymax), tzmax);
}


__device__ __forceinline__ 
void get_min_pos(const Vector3f_da& volume_hsize, const Vector3f_da& pos, const Vector3f_da& direction, Vector3f_da& min_voxel_pos)
{
    if (direction[0] < EPSILON && direction[0] > - EPSILON)
    {
        min_voxel_pos[0] = - volume_hsize[0];
    }
    else
    {
        min_voxel_pos[0] = ((direction[0] > 0 ? volume_hsize[0] : - volume_hsize[0]) - pos[0]) / direction[0];
    }
    if (direction[1] < EPSILON && direction[1] > - EPSILON)
    {
        min_voxel_pos[1] = - volume_hsize[1];
    }
    else
    {
        min_voxel_pos[1] = ((direction[1] > 0 ? volume_hsize[1] : - volume_hsize[1]) - pos[1]) / direction[1];
    }
    if (direction[2] < EPSILON && direction[2] > - EPSILON)
    {
        min_voxel_pos[2] = - volume_hsize[2];
    }
    else
    {
        min_voxel_pos[2] = ((direction[2] > 0 ? volume_hsize[2] : - volume_hsize[2]) - pos[2]) / direction[2];
    }
}


__device__ __forceinline__ 
void get_max_pos(const Vector3f_da& volume_hsize, const Vector3f_da& pos, const Vector3f_da& direction, Vector3f_da& max_voxel_pos)
{
    if (direction[0] < EPSILON && direction[0] > - EPSILON)
    {
        max_voxel_pos[0] = volume_hsize[0];
    }
    else
    {
        max_voxel_pos[0] = ((direction[0] > 0 ? - volume_hsize[0] : volume_hsize[0]) - pos[0]) / direction[0];
    }
    if (direction[1] < EPSILON && direction[1] > - EPSILON)
    {
        max_voxel_pos[1] = - volume_hsize[1];
    }
    else
    {
        max_voxel_pos[1] = ((direction[1] > 0 ? - volume_hsize[1] : volume_hsize[1]) - pos[1]) / direction[1];
    }
    if (direction[2] < EPSILON && direction[2] > - EPSILON)
    {
        max_voxel_pos[2] = - volume_hsize[2];
    }
    else
    {
        max_voxel_pos[2] = ((direction[2] > 0 ? - volume_hsize[2] : volume_hsize[2]) - pos[2]) / direction[2];
    }
}


__global__
void kernel_raycast_tsdf(
    const PtrStep<short2> tsdf_volume,
    const CameraParameters cam, const Matrix3f_da R_w_c, const Vector3f_da t_w_c,
    const int3 volume_size, const float voxel_scale, const float truncation_distance,
    PtrStepSz<float3> vertex_map, PtrStep<float3> normal_map
) 
{
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= vertex_map.cols || y >= vertex_map.rows) return;

    const float3 volume_range = make_float3(
        volume_size.x * voxel_scale, volume_size.y * voxel_scale, volume_size.z * voxel_scale
    );

    const Vector3f_da offset(volume_size.x / 2.f * voxel_scale, volume_size.y / 2.f * voxel_scale, volume_size.z / 2.f * voxel_scale);

    const Vector3f_da ray_c((x - cam.cx) / cam.fx, (y - cam.cy) / cam.fy, 1.f);  // in camera coordinate
    const Vector3f_da ray_direction = (R_w_c * ray_c).normalized();  // in world coordinate

    float min_length = 0.f;
    const Vector3f_da voxel_w = t_w_c + offset;

    Vector3f_da min_voxel_pos, max_voxel_pos;
    get_min_pos(offset, t_w_c, ray_direction, min_voxel_pos);
    get_max_pos(offset, t_w_c, ray_direction, max_voxel_pos);
    
    float ray_length = fmax(get_min_time(volume_range, t_w_c + offset, ray_direction), 0.f);
    const float max_length = get_max_time(volume_range, t_w_c + offset, ray_direction);
    if (ray_length >= max_length) return;

    ray_length += voxel_scale / 2.f;

    Vector3f_da grid = (t_w_c + ray_direction * ray_length + offset) / voxel_scale;

    if (grid[0] < 0 || grid[0] > volume_size.x - 1 ||
        grid[1] < 0 || grid[1] > volume_size.y - 1 ||
        grid[2] < 0 || grid[2] > volume_size.z - 1) return;
    
    float tsdf = static_cast<float>(
        tsdf_volume.ptr(__float2int_rd(grid[2]) * volume_size.y + __float2int_rd(grid[1]))[__float2int_rd(grid[0])].x
    ) * INV_SHORT_MAX;
    if (tsdf < 0.f) return;

    const float max_search_length = max_length;
    for (; ray_length < max_search_length; ray_length += truncation_distance * 0.5f)
    {
        grid += ray_direction * truncation_distance * 0.5f / voxel_scale;

        if (grid[0] < 0 || grid[0] > volume_size.x - 1 ||
            grid[1] < 0 || grid[1] > volume_size.y - 1 ||
            grid[2] < 0 || grid[2] > volume_size.z - 1) continue;

        const float previous_tsdf = tsdf;
        tsdf = static_cast<float>(
            tsdf_volume.ptr(__float2int_rd(grid[2]) * volume_size.y + __float2int_rd(grid[1]))[__float2int_rd(grid[0])].x
        ) * INV_SHORT_MAX;

        if (previous_tsdf < 0.f) return;
        if (previous_tsdf > 0.f && tsdf < 0.f)
        {
            //Zero crossing
            const float t_star = ray_length - truncation_distance * 0.5f * previous_tsdf / (tsdf - previous_tsdf);
            const auto vertex_w = t_w_c + ray_direction * t_star;

            const Vector3f_da location_in_grid = (vertex_w + offset) / voxel_scale;
            if (location_in_grid[0] < 1 || location_in_grid[0] >= volume_size.x - 1 ||
                location_in_grid[1] < 1 || location_in_grid[1] >= volume_size.y - 1 ||
                location_in_grid[2] < 1 || location_in_grid[2] >= volume_size.z - 1) return;
            
            Vector3f_da normal_w, shifted;

            shifted = location_in_grid;
            shifted[0] += 1;
            if (shifted[0] >= volume_size.x - 1) break;
            const float Fx1 = interpolate_trilinearly(shifted, tsdf_volume, volume_size, voxel_scale);

            shifted = location_in_grid;
            shifted.x() -= 1;
            if (shifted.x() < 1) break;
            const float Fx2 = interpolate_trilinearly(shifted, tsdf_volume, volume_size, voxel_scale);

            normal_w[0] = (Fx1 - Fx2);

            shifted = location_in_grid;
            shifted.y() += 1;
            if (shifted.y() >= volume_size.y - 1) break;
            const float Fy1 = interpolate_trilinearly(shifted, tsdf_volume, volume_size, voxel_scale);

            shifted = location_in_grid;
            shifted.y() -= 1;
            if (shifted.y() < 1) break;
            const float Fy2 = interpolate_trilinearly(shifted, tsdf_volume, volume_size, voxel_scale);

            normal_w[1] = (Fy1 - Fy2);

            shifted = location_in_grid;
            shifted.z() += 1;
            if (shifted.z() >= volume_size.z - 1) break;
            const float Fz1 = interpolate_trilinearly(shifted, tsdf_volume, volume_size, voxel_scale);

            shifted = location_in_grid;
            shifted.z() -= 1;
            if (shifted.z() < 1) break;
            const float Fz2 = interpolate_trilinearly(shifted, tsdf_volume, volume_size, voxel_scale);

            normal_w[2] = (Fz1 - Fz2);

            if (normal_w.norm() < EPSILON) break;

            normal_w.normalize();

            vertex_map.ptr(y)[x] = make_float3(vertex_w.x(), vertex_w.y(), vertex_w.z());
            normal_map.ptr(y)[x] = make_float3(normal_w.x(), normal_w.y(), normal_w.z());
            break;
        }
    }
}


void raycastTSDF(
    const TSDFData &tsdf_data,
    const CameraParameters &cam,
    const Eigen::Matrix4f &T_w_c,
    GpuMat &vertex_map, GpuMat &normal_map
)
{
    vertex_map.setTo(0);
    normal_map.setTo(0);

    const dim3 threads(32, 32);
    const dim3 blocks(divUp(vertex_map.cols, threads.x), divUp(vertex_map.rows, threads.y));

    kernel_raycast_tsdf<<<blocks, threads>>>(
        tsdf_data.tsdf, cam, 
        T_w_c.block<3, 3>(0, 0), T_w_c.block<3, 1>(0, 3),
        tsdf_data.volume_size, tsdf_data.voxel_scale,
        tsdf_data.truncation_distance,
        vertex_map, normal_map
    );

    hipDeviceSynchronize();
}
