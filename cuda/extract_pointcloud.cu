#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <opencv2/core/cuda.hpp>
#include <opencv2/cudev/common.hpp>
#include <Eigen/Core>
#include "datatypes.hpp"

using cv::cuda::PtrStepSz;
using Vector2i_da = Eigen::Matrix<int, 2, 1, Eigen::DontAlign>;
using Vector3f_da = Eigen::Matrix<float, 3, 1, Eigen::DontAlign>;
using Matrix3f_da = Eigen::Matrix<float, 3, 3, Eigen::DontAlign>;
constexpr float DIVSHORTMAX = 0.0000305185f;  // 1.f / SHORT_MAX;

__global__
void extract_points_kernel(
    const PtrStepSz<short2> tsdf_volume, 
    const int3 volume_size, const float voxel_scale,
    PtrStepSz<float3> vertices, PtrStepSz<float3> normals,
    int *point_num
) {
    const int x = blockIdx.x * blockDim.x + threadIdx.x;
    const int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= volume_size.x - 1 || y >= volume_size.y - 1) return;

    for (int z = 0; z < volume_size.z - 1; ++z) {
        const short2 value = tsdf_volume.ptr(z * volume_size.y + y)[x];

        const float tsdf = static_cast<float>(value.x) * DIVSHORTMAX;
        if (tsdf == 0 || tsdf <= -0.99f || tsdf >= 0.99f) continue;

        short2 vx = tsdf_volume.ptr((z) * volume_size.y + y)[x + 1];
        short2 vy = tsdf_volume.ptr((z) * volume_size.y + y + 1)[x];
        short2 vz = tsdf_volume.ptr((z + 1) * volume_size.y + y)[x];

        if (vx.y <= 0 || vy.y <= 0 || vz.y <= 0) continue;

        const float tsdf_x = static_cast<float>(vx.x) * DIVSHORTMAX;
        const float tsdf_y = static_cast<float>(vy.x) * DIVSHORTMAX;
        const float tsdf_z = static_cast<float>(vz.x) * DIVSHORTMAX;

        const bool is_surface_x = ((tsdf > 0) && (tsdf_x < 0)) || ((tsdf < 0) && (tsdf_x > 0));
        const bool is_surface_y = ((tsdf > 0) && (tsdf_y < 0)) || ((tsdf < 0) && (tsdf_y > 0));
        const bool is_surface_z = ((tsdf > 0) && (tsdf_z < 0)) || ((tsdf < 0) && (tsdf_z > 0));

        if (is_surface_x || is_surface_y || is_surface_z) {
            Eigen::Vector3f normal;
            normal.x() = (tsdf_x - tsdf);
            normal.y() = (tsdf_y - tsdf);
            normal.z() = (tsdf_z - tsdf);
            if (normal.norm() == 0) continue;
            normal.normalize();

            int count = 0;
            if (is_surface_x) count++;
            if (is_surface_y) count++;
            if (is_surface_z) count++;
            int index = atomicAdd(point_num, count);

            /*
            Vector3f_da position(
                (static_cast<float>(x) + 0.5f - volume_size.x / 2.f) * voxel_scale,
                (static_cast<float>(y) + 0.5f - volume_size.y / 2.f) * voxel_scale,
                (static_cast<float>(z) + 0.5f - volume_size.z / 2.f) * voxel_scale
            );
            */

            Vector3f_da position(
                (static_cast<float>(x) + 0.5f) * voxel_scale,
                (static_cast<float>(y) + 0.5f) * voxel_scale,
                (static_cast<float>(z) + 0.5f) * voxel_scale
            );
            if (is_surface_x) {
                position.x() = position.x() - (tsdf / (tsdf_x - tsdf)) * voxel_scale;

                vertices.ptr(0)[index] = float3{position(0), position(1), position(2)};
                normals.ptr(0)[index] = float3{normal(0), normal(1), normal(2)};
                index++;
            }
            if (is_surface_y) {
                position.y() -= (tsdf / (tsdf_y - tsdf)) * voxel_scale;

                vertices.ptr(0)[index] = float3{position(0), position(1), position(2)};;
                normals.ptr(0)[index] = float3{normal(0), normal(1), normal(2)};
                index++;
            }
            if (is_surface_z) {
                position.z() -= (tsdf / (tsdf_z - tsdf)) * voxel_scale;

                vertices.ptr(0)[index] = float3{position(0), position(1), position(2)};;
                normals.ptr(0)[index] = float3{normal(0), normal(1), normal(2)};
                index++;
            }
        }
    }
}

PointCloud extract_points(const TSDFData& volume, const int buffer_size)
{
    CloudData cloud_data { buffer_size };

    dim3 threads(32, 32);
    dim3 blocks(
        (volume.volume_size.x + threads.x - 1) / threads.x,
        (volume.volume_size.y + threads.y - 1) / threads.y
    );

    extract_points_kernel<<<blocks, threads>>>(
        volume.tsdf,
        volume.volume_size, volume.voxel_scale,
        cloud_data.vertices, cloud_data.normals, cloud_data.point_num
    );

    hipDeviceSynchronize();

    return cloud_data.download();
}
