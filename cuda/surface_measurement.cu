#include "hip/hip_runtime.h"
#include <opencv2/core/cuda.hpp>
#include <datatypes.hpp>


using cv::cuda::GpuMat;


__global__ void kernel_compute_vertex_map(const cv::cuda::PtrStepSz<float> depth_map, cv::cuda::PtrStepSz<float3> vertex_map, const CameraIntrinsics camera_params, const float max_depth){
    // Calculate global row and column for each thread
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col >= depth_map.cols || row >= depth_map.rows)
        return;

    float depth_val = depth_map(row, col);

    // Don't use depth values larger than max_depth
    if (depth_val > max_depth){
        depth_val = 0.f;
    } 

    // from screen to camera space
    vertex_map(row, col) = make_float3((col - camera_params.cx) * depth_val / camera_params.fx,
                                       (row - camera_params.cy) * depth_val / camera_params.fy,
                                       depth_val);
}

__global__ void kernel_compute_normal_map(cv::cuda::PtrStepSz<float3> vertex_map, cv::cuda::PtrStepSz<float3> normal_map){
    // Calculate global row and column for each thread
    const int col = blockIdx.x * blockDim.x + threadIdx.x;
    const int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col >= vertex_map.cols - 1 || row >= vertex_map.rows - 1){
        if (col == vertex_map.cols - 1 || row == vertex_map.rows - 1){
            normal_map(row, col) = make_float3(0.f,0.f,0.f);            // TODO: maybe compute them with vertex_map(row - 1, col) etc.
        }
        return;
    }
    float sx = vertex_map(row + 1, col).x - vertex_map(row, col).x;
    float sy = vertex_map(row + 1, col).y - vertex_map(row, col).y;
    float sz = vertex_map(row + 1, col).z - vertex_map(row, col).z;
    float tx = vertex_map(row, col + 1).x - vertex_map(row, col).x;
    float ty = vertex_map(row, col + 1).y - vertex_map(row, col).y;
    float tz = vertex_map(row, col + 1).z - vertex_map(row, col).z;

    float3 cross_prod = make_float3(sy * tz - sz * ty, 
                                    sz * tx - sx * tz,
                                    sx * ty - sy * tx);

    float norm = sqrt(cross_prod.x * cross_prod.x + cross_prod.y * cross_prod.y + cross_prod.z * cross_prod.z) + .000001f;

    normal_map(row, col) = make_float3(cross_prod.x / norm,
                                       cross_prod.y / norm,
                                       cross_prod.z / norm);
}

void compute_vertex_map(const GpuMat& filtered_depth_map, GpuMat& vertex_map, const CameraIntrinsics camera_params, const float max_depth){
    int threads = 32;
    dim3 T(threads, threads, 1);      // number of threads per block (depends on compute capability of your GPU)
    int blocks_x = (filtered_depth_map.cols + T.x - 1) / T.x;
    int blocks_y = (filtered_depth_map.rows + T.y - 1) / T.y;
    dim3 M(blocks_x, blocks_y, 1);       // number of thread blocks (depends on compute capability of your GPU)
    kernel_compute_vertex_map<<< M , T >>>(filtered_depth_map, vertex_map, camera_params, max_depth);
    hipDeviceSynchronize();
}

void compute_normal_map(const GpuMat& vertex_map, GpuMat& normal_map){
    int threads = 32;
    dim3 T(threads, threads, 1);      // number of threads per block (depends on compute capability of your GPU)
    int blocks_x = (vertex_map.cols + T.x - 1) / T.x;
    int blocks_y = (vertex_map.rows + T.y - 1) / T.y;
    dim3 M(blocks_x, blocks_y, 1);       // number of thread blocks (depends on compute capability of your GPU)
    kernel_compute_normal_map<<< M , T >>>(vertex_map, normal_map);
    hipDeviceSynchronize();
}

